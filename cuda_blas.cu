#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>


/**
 * Main function.
 */
int main(int argc, char **argv) {
  int length;
  float *h_x=NULL, *h_y=NULL;
  float y_true = NAN;

  float *d_x=NULL, *d_y=NULL;
  hipEvent_t event_begin, event_end;
  float elapsed_time_ms=0;
  int n_threads=-1;
  const float a=1.0;
  hipblasHandle_t cublasH=NULL;

  /* read problem size/length from command line */
  if (2 <= argc) {
    length = strtol(argv[1], NULL, 10);
  }
  printf("Problem size: length=%g\n", (double) length);
  fflush(stdout); // flush buffered print output

  /* allocate arrays in host memory */
  h_x = (float*) malloc(length*sizeof(float));
  h_y = (float*) malloc(length*sizeof(float));

  /* initialize x and y arrays */
  for (int i=0; i<length; i++) {
    h_x[i] = 1.3;
    h_y[i] = 2.4;
  }

  /* set reference solution */
  y_true = 2.4 + 11*1.0*1.3;
  /* create CUDA events for timing */
  hipEventCreate(&event_begin);
  hipEventCreate(&event_end);
  /* create cuBLAS handle */
  hipblasCreate(&cublasH);
  /* allocate arrays in device memory */
  hipMalloc(&d_x, length*sizeof(float));
  hipMalloc(&d_y, length*sizeof(float));

  printf("========================================\n");
  fflush(stdout); // flush buffered print outputs

  /* transfer data from host to device memory */
  hipMemcpy(d_x, h_x, length*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, length*sizeof(float), hipMemcpyHostToDevice);
  /* warm-up run */
  hipblasSaxpy(cublasH, length, &a, d_x, 1, d_y, 1);
  /* begin timing */
  hipEventRecord(event_begin, 0);
  /* timing run */
  for (int k=1; k<11; k++) {
    hipblasSaxpy(cublasH, length, &a, d_x, 1, d_y, 1);
  }
  /* wait for GPU threads to complete */
  hipDeviceSynchronize();
  /* end timing */
  hipEventRecord(event_end, 0);
  hipEventSynchronize(event_end);
  hipEventElapsedTime(&elapsed_time_ms, event_begin, event_end);
  /* transfer data from device to host memory */
  hipMemcpy(h_y, d_y, length*sizeof(float), hipMemcpyDeviceToHost);
  /* print timing result */
  printf("cublasSaxpy:\t"
         "Number of threads %d,\tWall-clock time [ms] ~ %g\n",
         n_threads, elapsed_time_ms);

  /* check errors */
  float max_error = 0.0;
  for (int i=0; i<length; i++) {
    max_error = fmax(max_error, fabs(h_y[i] - y_true));
  }
  printf("Max error = %e\n", max_error);

  printf("========================================\n");
  fflush(stdout); // flush buffered print outputs

  /* deallocate arrays in device memory */
  hipFree(d_x);
  hipFree(d_y);

  /* deallocate arrays in host memory */
  free(h_x);
  free(h_y);

  return 0;
}